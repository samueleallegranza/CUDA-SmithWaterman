
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define S_LEN 32
#define N 4

#define PEN_INS -2
#define PEN_DEL -2
#define PEN_MATCH 1
#define PEN_MISMATCH -1

#define VAL 0
#define J   1

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

double get_time() // function that returns the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

__device__ int max4(int n1, int n2, int n3, int n4)
{
	int tmp1, tmp2;
	tmp1 = n1 > n2 ? n1 : n2;
	tmp2 = n3 > n4 ? n3 : n4;
	tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
	return tmp1;
}

__device__ void backtrace(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
	int n;
	for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
	{
		int dir = dir_mat[i][j];
		if (dir == 1 || dir == 2)
		{
			i--;
			j--;
		}
		else if (dir == 3)
			i--;
		else if (dir == 4)
			j--;

		simple_rev_cigar[n] = dir;
	}
}

__device__ int setDirScore(char * query, char * reference, int ** sc_mat, char ** dir_mat, int x, int y) {
	// penalities (initialized above)
	// 	ins = -2
	// 	del = -2
	// 	match = 1
	// 	mismatch = -1 

	// compare the sequences characters
	int comparison = (query[x - 1] == reference[y - 1]) ? PEN_MATCH : PEN_MISMATCH;
	// compute the cell knowing the comparison resultS
	int score = max4(sc_mat[x - 1][y - 1] + comparison, sc_mat[x - 1][y] + PEN_DEL, sc_mat[x][y- 1] + PEN_INS, 0);

	// directions
	// 	1 -> up + left
	// 	2 -> up + left
	// 	3 -> left
	// 	4 -> up
	//  0 -> none.
	char dir;

	if (score == (sc_mat[x - 1][y - 1] + comparison))
		dir = comparison == PEN_MATCH ? 1 : 2;
	else if (score == (sc_mat[x - 1][y] + PEN_DEL))
		dir = 3;
	else if (score == (sc_mat[x][y - 1] + PEN_INS))
		dir = 4;
	else
		dir = 0;

	dir_mat[x][y] = dir;
	sc_mat[x][y] = score;

	return score;
}

__device__ void sw_block_calc(char * query, char * reference, int ** sc_mat, char ** dir_mat, int start_x, int start_y, int side_len, int threadBlockIdx) {    
	int i, j, j_end;
	int progress, progress_tot;

    i = start_y + threadBlockIdx;
    j = start_x;
	j_end = start_x + side_len - 1;
	progress_tot = (side_len*2)-1;
	for(progress=0; progress < progress_tot; progress++) {
		if(threadBlockIdx <= progress && j <= j_end && (i < S_LEN && j < S_LEN)) {
			setDirScore(query, reference, sc_mat, dir_mat, i+1, j+1);
			j++;
		}
        __syncthreads();
    }
}

__global__ void kernel_launch(char ** query, char ** reference, int *** sc_mat, char *** dir_mat, int * res, char ** simple_rev_cigar, int blocks_per_area, int side_blocks) {
	int areaIdx, blockAreaIdx, threadBlockIdx;
	int i, margin;
	int start_x, start_y;

	areaIdx = (int) blockIdx.x / blocks_per_area;
	blockAreaIdx = blockIdx.x % blocks_per_area;
	threadBlockIdx = threadIdx.x;

	start_x = (blockAreaIdx % side_blocks) * blockDim.x;
	start_y = ((int) blockAreaIdx / side_blocks) * blockDim.x;

	if(blockAreaIdx == 0 && threadBlockIdx == 0) {
		for(int k=0; k<S_LEN+1; k++) {
			dir_mat[areaIdx][0][k] = 0;
			sc_mat[areaIdx][0][k] = 0;
			dir_mat[areaIdx][k][0] = 0;
			sc_mat[areaIdx][k][0] = 0;
		}
	}

	// if(areaIdx == 0 && threadBlockIdx == 0) {
	// 	printf("[BL %d][TH %d] start_x=%d; start_y=%d\n", blockAreaIdx, threadBlockIdx, start_x, start_y);
	// }

	margin = 0;
	for(i=0; i<side_blocks; i++) {
		if(((blockAreaIdx-i)%(side_blocks-1))==0 && blockAreaIdx <= margin && blockAreaIdx >= i) {
			if(areaIdx == 0 && threadBlockIdx == 0) {printf("[DIAG IDX %d] [BL %d][TH %d] start_x=%d; start_y=%d\n", i, blockAreaIdx, threadBlockIdx, start_x, start_y);}
			sw_block_calc(query[areaIdx], reference[areaIdx], sc_mat[areaIdx], dir_mat[areaIdx], start_x, start_y, side_blocks, threadBlockIdx);
		}
		margin += side_blocks;
		__syncthreads();
	}
}


__host__ int max4_CPU(int n1, int n2, int n3, int n4)
{
	int tmp1, tmp2;
	tmp1 = n1 > n2 ? n1 : n2;
	tmp2 = n3 > n4 ? n3 : n4;
	tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
	return tmp1;
}

__host__ void backtrace_CPU(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
	int n;
	for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
	{
		int dir = dir_mat[i][j];
		if (dir == 1 || dir == 2)
		{
			i--;
			j--;
		}
		else if (dir == 3)
			i--;
		else if (dir == 4)
			j--;

		simple_rev_cigar[n] = dir;
	}
}

__host__ void sw_CPU(char ** query, char ** reference, int ** sc_mat, char ** dir_mat, int * res, char ** simple_rev_cigar) {
	int ins = -2, del = -2, match = 1, mismatch = -1; // penalties
	
	for (int n = 0; n < N; n++)
	{
		int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
		int maxi, maxj;
		// initialize the scoring matrix and direction matrix to 0
		for (int i = 0; i < S_LEN + 1; i++)
		{
			for (int j = 0; j < S_LEN + 1; j++)
			{
				sc_mat[i][j] = 0;
				dir_mat[i][j] = 0;
			}
		}
		// compute the alignment
		for (int i = 1; i < S_LEN + 1; i++)
		{
			for (int j = 1; j < S_LEN + 1; j++)
			{
				// compare the sequences characters
				int comparison = (query[n][i - 1] == reference[n][j - 1]) ? match : mismatch;
				// compute the cell knowing the comparison result
				int tmp = max4_CPU(sc_mat[i - 1][j - 1] + comparison, sc_mat[i - 1][j] + del, sc_mat[i][j - 1] + ins, 0);
				char dir;

				if (tmp == (sc_mat[i - 1][j - 1] + comparison))
					dir = comparison == match ? 1 : 2;
				else if (tmp == (sc_mat[i - 1][j] + del))
					dir = 3;
				else if (tmp == (sc_mat[i][j - 1] + ins))
					dir = 4;
				else
					dir = 0;

				dir_mat[i][j] = dir;
				sc_mat[i][j] = tmp;

				if (tmp > max)
				{
					max = tmp;
					maxi = i;
					maxj = j;
				}
			}
		}
		res[n] = sc_mat[maxi][maxj];
		backtrace_CPU(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN * 2);

        // if(n == 0) {
        //     printf("Score matrix\t\tDirection matrix\n");
        //     for(int i=0; i<S_LEN; i++) {
        //         for(int j=0; j<S_LEN+1; j++) {
        //             printf("%d ", sc_mat[i][j]);
        //         }
        //         printf("\t");
        //         for(int j=0; j<S_LEN+1; j++) {
        //             printf("%d ", dir_mat[i][j]);
        //         }
        //         printf("\n");
        //     }
        //     printf("max = %d, maxi = %d, maxj = %d\n", max, maxi, maxj);
        // }
    }
}

int main(int argc, char * argv[]) {
	srand(time(NULL)); 
    char alphabet[5] = {'A', 'C', 'G', 'T', 'N'};
    double time_start, time_stop;
    int BLOCKSIZE = atoi(argv[1]);

    // Host memory allocation and initialization for sequences (randomly generated)
    char **h_query = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_query[i] = (char *)malloc(S_LEN * sizeof(char));
	char **h_reference = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_reference[i] = (char *)malloc(S_LEN * sizeof(char));

	int **h_sc_mat = (int **)malloc((S_LEN + 1) * sizeof(int *));
	for (int i = 0; i < (S_LEN + 1); i++)
		h_sc_mat[i] = (int *)malloc((S_LEN + 1) * sizeof(int));
	char **h_dir_mat = (char **)malloc((S_LEN + 1) * sizeof(char *));
	for (int i = 0; i < (S_LEN + 1); i++)
		h_dir_mat[i] = (char *)malloc((S_LEN + 1) * sizeof(char));

	int *h_res = (int *)malloc(N * sizeof(int));
	char **h_simple_rev_cigar = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_simple_rev_cigar[i] = (char *)malloc(S_LEN * 2 * sizeof(char));


	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < S_LEN; j++)
		{
			h_query[i][j] = alphabet[rand() % 5];
			h_reference[i][j] = alphabet[rand() % 5];
		}
	}

    // Device memory allocation and transfer of input data
    char **d_query;
    char **d_query_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_query,  N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_query_ptrs[i]), S_LEN*sizeof(char));
        hipMemcpy (d_query_ptrs[i], h_query[i], S_LEN*sizeof(char), hipMemcpyHostToDevice);
    }
    hipMemcpy (d_query, d_query_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);

    char **d_reference;
    char **d_reference_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_reference,  N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_reference_ptrs[i]), S_LEN*sizeof(char));
        hipMemcpy (d_reference_ptrs[i], h_reference[i], S_LEN*sizeof(char), hipMemcpyHostToDevice);
    }
    hipMemcpy (d_reference, d_reference_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);
	
	int **d_sc_mat;
    int **d_sc_mat_ptrs;
	int ***d_sc_mat_list;
	int ***d_sc_mat_list_ptrs = (int ***)malloc(N * sizeof(int **));
    hipMalloc((void***)&d_sc_mat_list, N * sizeof(int **));
	for(int j=0; j<N; j++) {
		d_sc_mat_ptrs = (int **)malloc((S_LEN+1) * sizeof(int *));    
		hipMalloc((void***)&d_sc_mat, (S_LEN+1) * sizeof(int *));
		for(int i=0; i<(S_LEN+1); i++) {
			hipMalloc((void**) &(d_sc_mat_ptrs[i]), (S_LEN+1)*sizeof(int));
		}
		hipMemcpy (d_sc_mat, d_sc_mat_ptrs, (S_LEN+1)*sizeof(int *), hipMemcpyHostToDevice);
		d_sc_mat_list_ptrs[j] = d_sc_mat;
	}
	hipMemcpy (d_sc_mat_list, d_sc_mat_list_ptrs, N*sizeof(int **), hipMemcpyHostToDevice);

	char **d_dir_mat;
    char **d_dir_mat_ptrs;
	char ***d_dir_mat_list;
	char ***d_dir_mat_list_ptrs = (char ***)malloc(N * sizeof(char **));
    hipMalloc((void***)&d_dir_mat_list, N * sizeof(char **));
	for(int j=0; j<N; j++) {
		d_dir_mat_ptrs = (char **)malloc((S_LEN+1) * sizeof(char *));    
		hipMalloc((void***)&d_dir_mat, (S_LEN+1) * sizeof(char *));
		for(int i=0; i<(S_LEN+1); i++) {
			hipMalloc((void**) &(d_dir_mat_ptrs[i]), (S_LEN+1)*sizeof(char));
		}
		hipMemcpy (d_dir_mat, d_dir_mat_ptrs, (S_LEN+1)*sizeof(char *), hipMemcpyHostToDevice);
		d_dir_mat_list_ptrs[j] = d_dir_mat;
	}
	hipMemcpy (d_dir_mat_list, d_dir_mat_list_ptrs, N*sizeof(char **), hipMemcpyHostToDevice);

    int * d_res;
    hipMalloc((void**) &d_res, N*sizeof(int));

    char **d_simple_rev_cigar;
    char **d_simple_rev_cigar_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_simple_rev_cigar, N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_simple_rev_cigar_ptrs[i]), S_LEN * 2 * sizeof(char));
    }
    hipMemcpy (d_simple_rev_cigar, d_simple_rev_cigar_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);

    // Blocks and threads schema for GPU execution
	int side_blocks = (int) ceil((float) S_LEN / BLOCKSIZE);
	int blocks_per_area = side_blocks * side_blocks;
	dim3 blocksPerGrid(blocks_per_area*N, 1, 1);
    dim3 threadsPerBlock(BLOCKSIZE, 1, 1);
	printf("BLOCKSIZE = %d;  blocks_per_area = %d\n", BLOCKSIZE, blocks_per_area);

    // Execution on GPU started
    time_start = get_time();

    kernel_launch<<<blocksPerGrid, threadsPerBlock>>>(d_query, d_reference, d_sc_mat_list, d_dir_mat_list, d_res, d_simple_rev_cigar, blocks_per_area, side_blocks);
    CHECK_KERNELCALL();
    hipDeviceSynchronize();

    // Execution on GPU ended
    time_stop = get_time();
    printf("GPU Execution time: %.10f\n", time_stop-time_start);

	// Transfer data to host
	// int * gpu_res = (int *) malloc(sizeof(int)*N);
	int gpu_res[N];
	char gpu_simple_rev_cigar[N][S_LEN*2];
	char * tmp_pointer;

	hipMemcpy((void *) gpu_res, d_res, sizeof(int)*N, hipMemcpyDeviceToHost);
	for(int i=0; i<N; i++) {
		tmp_pointer = d_simple_rev_cigar_ptrs[i];
		hipMemcpy((void *) gpu_simple_rev_cigar[i], tmp_pointer, sizeof(char)*S_LEN*2, hipMemcpyDeviceToHost);
	}

    // Execution on CPU started
    time_start = get_time();

	sw_CPU(h_query, h_reference, h_sc_mat, h_dir_mat, h_res, h_simple_rev_cigar);
    
	// Execution on CPU ended
    time_stop = get_time();
    printf("CPU Execution time: %.10f [!]\n", time_stop-time_start);

	// Check if results are consistent
	// "Results" validation
	int ok, i, j;
	ok = 1;
	for(i=0; (i<N && ok); i++) {
		ok = gpu_res[i] == h_res[i];
	}

	if(ok)
		printf("[OK]\t'results' is consistent\n");
	else
		printf("[ERR]\t'results' is inconsistent\n");

	ok = 1;
	for(i=0; (i<N && ok); i++) {
		for(j=0; (j<S_LEN*2 && ok); j++) {
			ok = gpu_simple_rev_cigar[i][j] == h_simple_rev_cigar[i][j];
		}
	}
	i--;
	if(ok)
		printf("[OK]\t'rev_cigar' is consistent\n");
	else{
		printf("[ERR]\t'rev_cigar' is inconsistent [err on: %d]\n", i);
	}


	// Deallocation of memory
	CHECK(hipFree(d_query));
    CHECK(hipFree(d_reference));
    CHECK(hipFree(d_sc_mat_list));
    CHECK(hipFree(d_dir_mat_list));
    CHECK(hipFree(d_res));
    CHECK(hipFree(d_simple_rev_cigar));

	free(h_query);
	free(h_reference);
	free(h_sc_mat);
	free(h_dir_mat);
	free(h_res);
	free(h_simple_rev_cigar);

    return 0;
}