
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define S_LEN 512
#define N 1000

#define PEN_INS -2
#define PEN_DEL -2
#define PEN_MATCH 1
#define PEN_MISMATCH -1

#define D 0
#define X 1
#define Y 2

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

typedef struct max_supp_s {
	int v[3];
	int i[3];
	int j[3];
} max_supp_t;

double get_time() // function that returns the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

__device__ int max4(int n1, int n2, int n3, int n4)
{
	int tmp1, tmp2;
	tmp1 = n1 > n2 ? n1 : n2;
	tmp2 = n3 > n4 ? n3 : n4;
	tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
	return tmp1;
}

__device__ void backtrace(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
	int n;
	for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
	{
		int dir = dir_mat[i][j];
		if (dir == 1 || dir == 2)
		{
			i--;
			j--;
		}
		else if (dir == 3)
			i--;
		else if (dir == 4)
			j--;

		simple_rev_cigar[n] = dir;
	}
}

__device__ int setDirScore(char * query, char * reference, int ** sc_mat, char ** dir_mat, int x, int y) {
	// penalities (initialized above)
	// 	ins = -2
	// 	del = -2
	// 	match = 1
	// 	mismatch = -1 

	// compare the sequences characters
	int comparison = (query[x - 1] == reference[y - 1]) ? PEN_MATCH : PEN_MISMATCH;
	// compute the cell knowing the comparison resultS
	int score = max4(sc_mat[x - 1][y - 1] + comparison, sc_mat[x - 1][y] + PEN_DEL, sc_mat[x][y- 1] + PEN_INS, 0);

	// directions
	// 	1 -> up + left
	// 	2 -> up + left
	// 	3 -> left
	// 	4 -> up
	//  0 -> none.
	char dir;

	if (score == (sc_mat[x - 1][y - 1] + comparison))
		dir = comparison == PEN_MATCH ? 1 : 2;
	else if (score == (sc_mat[x - 1][y] + PEN_DEL))
		dir = 3;
	else if (score == (sc_mat[x][y - 1] + PEN_INS))
		dir = 4;
	else
		dir = 0;

	dir_mat[x][y] = dir;
	sc_mat[x][y] = score;

	return score;
}

__global__ void sw_GPU(char ** query, char ** reference, int *** sc_mat_list, char *** dir_mat_list, int * res, char ** simple_rev_cigar, max_supp_t * max_supp) {    
	// srand(time(NULL)); 
	int n = blockIdx.x;
	
	int ** sc_mat = sc_mat_list[n];
	char ** dir_mat = dir_mat_list[n];

	int score;
	int max, maxi, maxj;

	// max_supp_t max_tmp = max_supp[n];
    __shared__ max_supp_t max_tmp;  

	// initialize the scoring matrix and direction matrix to 0 
    if(threadIdx.x == 0) {
        max_tmp.v[D] = PEN_INS;
        max_tmp.v[X] = PEN_INS;
        max_tmp.v[Y] = PEN_INS;

        for (int i = 0; i < S_LEN + 1; i++) {
            for (int j = 0; j < S_LEN + 1; j++) {
                sc_mat[i][j] = 0;
                dir_mat[i][j] = 0;
            }
        }

        // Necessary initialization for vertical comparison of scores
        max_tmp.i[Y] = INT_MAX;
        max_tmp.j[Y] = INT_MAX;
    }

	// compute the alignment
	for (int i = 1; i < S_LEN; i++) {
		// Handle diagonal cells
		if(threadIdx.x == 0) {
            score = setDirScore(query[n], reference[n], sc_mat, dir_mat, i, i);
            if(score > max_tmp.v[D]) {
                max_tmp.v[D] = score; max_tmp.i[D] = i; max_tmp.j[D] = i;
            }
        }
		// Handle vertical/horizontal cells
        __syncthreads();
		for(int j = i+1; j < S_LEN; j++) {
            if(threadIdx.x == 0) {
                score = setDirScore(query[n], reference[n], sc_mat, dir_mat, i, j);
                if(score > max_tmp.v[X]) {
                    max_tmp.v[X] = score; max_tmp.i[X] = i; max_tmp.j[X] = j;
                }
            } else if(threadIdx.x == 1) {
                score = setDirScore(query[n], reference[n], sc_mat, dir_mat, j, i);
                if(score > max_tmp.v[Y] || (score == max_tmp.v[Y] && j<max_tmp.i[Y])) {
                    max_tmp.v[Y]= score; max_tmp.i[Y] = j; max_tmp.j[Y] = i;
                }
            }
		}
	}

    if(threadIdx.x == 0) {
        int eq_maxes_idx[3];
        int eq_maxes_count = 0;
        int min_ij_score = INT_MAX;
        int ij_score;
        max = PEN_INS;
        maxi = -1; maxj = -1;
        // Find maximum value
        for(int i=0; i<3; i++)
            if(max_tmp.v[i] > max)
                max = max_tmp.v[i];
        for(int i=0; i<3; i++)
            if(max_tmp.v[i] == max) {
                eq_maxes_idx[eq_maxes_count] = i;
                eq_maxes_count++;
            }
        for(int i=0; i<eq_maxes_count; i++) {
            ij_score = max_tmp.i[eq_maxes_idx[i]]*(S_LEN)+max_tmp.j[eq_maxes_idx[i]];
            if(ij_score < min_ij_score) {
                min_ij_score = ij_score;
                maxi = max_tmp.i[eq_maxes_idx[i]];
                maxj = max_tmp.j[eq_maxes_idx[i]];
            }
        }

        res[n] = sc_mat[maxi][maxj];
        backtrace(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN * 2);
    }
}


__host__ int max4_CPU(int n1, int n2, int n3, int n4)
{
	int tmp1, tmp2;
	tmp1 = n1 > n2 ? n1 : n2;
	tmp2 = n3 > n4 ? n3 : n4;
	tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
	return tmp1;
}

__host__ void backtrace_CPU(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
	int n;
	for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
	{
		int dir = dir_mat[i][j];
		if (dir == 1 || dir == 2)
		{
			i--;
			j--;
		}
		else if (dir == 3)
			i--;
		else if (dir == 4)
			j--;

		simple_rev_cigar[n] = dir;
	}
}

__host__ void sw_CPU(char ** query, char ** reference, int ** sc_mat, char ** dir_mat, int * res, char ** simple_rev_cigar) {
	int ins = -2, del = -2, match = 1, mismatch = -1; // penalties
	
	for (int n = 0; n < N; n++)
	{
		int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
		int maxi, maxj;
		// initialize the scoring matrix and direction matrix to 0
		for (int i = 0; i < S_LEN + 1; i++)
		{
			for (int j = 0; j < S_LEN + 1; j++)
			{
				sc_mat[i][j] = 0;
				dir_mat[i][j] = 0;
			}
		}
		// compute the alignment
		for (int i = 1; i < S_LEN; i++)
		{
			for (int j = 1; j < S_LEN; j++)
			{
				// compare the sequences characters
				int comparison = (query[n][i - 1] == reference[n][j - 1]) ? match : mismatch;
				// compute the cell knowing the comparison result
				int tmp = max4_CPU(sc_mat[i - 1][j - 1] + comparison, sc_mat[i - 1][j] + del, sc_mat[i][j - 1] + ins, 0);
				char dir;

				if (tmp == (sc_mat[i - 1][j - 1] + comparison))
					dir = comparison == match ? 1 : 2;
				else if (tmp == (sc_mat[i - 1][j] + del))
					dir = 3;
				else if (tmp == (sc_mat[i][j - 1] + ins))
					dir = 4;
				else
					dir = 0;

				dir_mat[i][j] = dir;
				sc_mat[i][j] = tmp;

				if (tmp > max)
				{
					max = tmp;
					maxi = i;
					maxj = j;
				}
			}
		}
		res[n] = sc_mat[maxi][maxj];
		backtrace_CPU(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN * 2);
	}
}

int main(int argc, char * argv[]) {
	srand(time(NULL)); 
    char alphabet[5] = {'A', 'C', 'G', 'T', 'N'};
    double time_start, time_stop;

    // Host memory allocation and initialization for sequences (randomly generated)
    char **h_query = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_query[i] = (char *)malloc(S_LEN * sizeof(char));
	char **h_reference = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_reference[i] = (char *)malloc(S_LEN * sizeof(char));

	int **h_sc_mat = (int **)malloc((S_LEN + 1) * sizeof(int *));
	for (int i = 0; i < (S_LEN + 1); i++)
		h_sc_mat[i] = (int *)malloc((S_LEN + 1) * sizeof(int));
	char **h_dir_mat = (char **)malloc((S_LEN + 1) * sizeof(char *));
	for (int i = 0; i < (S_LEN + 1); i++)
		h_dir_mat[i] = (char *)malloc((S_LEN + 1) * sizeof(char));

	int *h_res = (int *)malloc(N * sizeof(int));
	char **h_simple_rev_cigar = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_simple_rev_cigar[i] = (char *)malloc(S_LEN * 2 * sizeof(char));


	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < S_LEN; j++)
		{
			h_query[i][j] = alphabet[rand() % 5];
			h_reference[i][j] = alphabet[rand() % 5];
		}
	}

    // Device memory allocation and transfer of input data
    char **d_query;
    char **d_query_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_query,  N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_query_ptrs[i]), S_LEN*sizeof(char));
        hipMemcpy (d_query_ptrs[i], h_query[i], S_LEN*sizeof(char), hipMemcpyHostToDevice);
    }
    hipMemcpy (d_query, d_query_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);

    char **d_reference;
    char **d_reference_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_reference,  N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_reference_ptrs[i]), S_LEN*sizeof(char));
        hipMemcpy (d_reference_ptrs[i], h_reference[i], S_LEN*sizeof(char), hipMemcpyHostToDevice);
    }
    hipMemcpy (d_reference, d_reference_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);
	
	int **d_sc_mat;
    int **d_sc_mat_ptrs;
	int ***d_sc_mat_list;
	int ***d_sc_mat_list_ptrs = (int ***)malloc(N * sizeof(int **));
    hipMalloc((void***)&d_sc_mat_list, N * sizeof(int **));
	for(int j=0; j<N; j++) {
		d_sc_mat_ptrs = (int **)malloc((S_LEN+1) * sizeof(int *));    
		hipMalloc((void***)&d_sc_mat, (S_LEN+1) * sizeof(int *));
		for(int i=0; i<(S_LEN+1); i++) {
			hipMalloc((void**) &(d_sc_mat_ptrs[i]), (S_LEN+1)*sizeof(int));
		}
		hipMemcpy (d_sc_mat, d_sc_mat_ptrs, (S_LEN+1)*sizeof(int *), hipMemcpyHostToDevice);
		d_sc_mat_list_ptrs[j] = d_sc_mat;
	}
	hipMemcpy (d_sc_mat_list, d_sc_mat_list_ptrs, N*sizeof(int **), hipMemcpyHostToDevice);

	char **d_dir_mat;
    char **d_dir_mat_ptrs;
	char ***d_dir_mat_list;
	char ***d_dir_mat_list_ptrs = (char ***)malloc(N * sizeof(char **));
    hipMalloc((void***)&d_dir_mat_list, N * sizeof(char **));
	for(int j=0; j<N; j++) {
		d_dir_mat_ptrs = (char **)malloc((S_LEN+1) * sizeof(char *));    
		hipMalloc((void***)&d_dir_mat, (S_LEN+1) * sizeof(char *));
		for(int i=0; i<(S_LEN+1); i++) {
			hipMalloc((void**) &(d_dir_mat_ptrs[i]), (S_LEN+1)*sizeof(char));
		}
		hipMemcpy (d_dir_mat, d_dir_mat_ptrs, (S_LEN+1)*sizeof(char *), hipMemcpyHostToDevice);
		d_dir_mat_list_ptrs[j] = d_dir_mat;
	}
	hipMemcpy (d_dir_mat_list, d_dir_mat_list_ptrs, N*sizeof(char **), hipMemcpyHostToDevice);

    int * d_res;
    hipMalloc((void**) &d_res, N*sizeof(int));

    char **d_simple_rev_cigar;
    char **d_simple_rev_cigar_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_simple_rev_cigar, N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_simple_rev_cigar_ptrs[i]), S_LEN * 2 * sizeof(char));
    }
    hipMemcpy (d_simple_rev_cigar, d_simple_rev_cigar_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);

    max_supp_t * d_max_supp;
    hipMalloc(&d_max_supp, N*sizeof(max_supp_t));


    // Blocks and threads schema for GPU execution
    dim3 blocksPerGrid(N, 1, 1);
    dim3 threadsPerBlock(2, 1, 1);

    // Execution on GPU started
    time_start = get_time();

    sw_GPU<<<blocksPerGrid, threadsPerBlock>>>(d_query, d_reference, d_sc_mat_list, d_dir_mat_list, d_res, d_simple_rev_cigar, d_max_supp);
    CHECK_KERNELCALL();
	// The following function adds A LOT of latency! Don't use it for benchmarking purposes
    // cudaDeviceSynchronize();

    // Execution on GPU ended
    time_stop = get_time();
    printf("GPU Execution time: %.10f\n", time_stop-time_start);

	// Transfer data to host
	// int * gpu_res = (int *) malloc(sizeof(int)*N);
	int gpu_res[N];
	char gpu_simple_rev_cigar[N][S_LEN*2];
	char * tmp_pointer;

	hipMemcpy((void *) gpu_res, d_res, sizeof(int)*N, hipMemcpyDeviceToHost);
	for(int i=0; i<N; i++) {
		tmp_pointer = d_simple_rev_cigar_ptrs[i];
		hipMemcpy((void *) gpu_simple_rev_cigar[i], tmp_pointer, sizeof(char)*S_LEN*2, hipMemcpyDeviceToHost);
	}

    // Execution on CPU started
    time_start = get_time();

	sw_CPU(h_query, h_reference, h_sc_mat, h_dir_mat, h_res, h_simple_rev_cigar);
    
	// Execution on CPU ended
    time_stop = get_time();
    printf("CPU Execution time: %.10f [!]\n", time_stop-time_start);

	// Check if results are consistent
	// "Results" validation
	int ok, i, j;
	ok = 1;
	for(i=0; (i<N && ok); i++) {
		ok = gpu_res[i] == h_res[i];
	}

	if(ok)
		printf("[OK]\t'results' is consistent\n");
	else
		printf("[ERR]\t'results' is inconsistent\n");

	ok = 1;
	for(i=0; (i<N && ok); i++) {
		for(j=0; (j<S_LEN*2 && ok); j++) {
			ok = gpu_simple_rev_cigar[i][j] == h_simple_rev_cigar[i][j];
		}
	}
	i--;
	if(ok)
		printf("[OK]\t'rev_cigar' is consistent\n");
	else{
		printf("[ERR]\t'rev_cigar' is inconsistent [err on: %d]\n", i);
		// printf("Comparison:\n");
		// for(j=0; j<(S_LEN*2); j++) {
		// 	printf("%d ", gpu_simple_rev_cigar[i][j]);
		// }
		// printf("\n");
		// for(j=0; j<(S_LEN*2); j++) {
		// 	printf("%d ", h_simple_rev_cigar[i][j]);
		// }
		// printf("\n");
	}


	// Deallocation of memory
	CHECK(hipFree(d_query));
    CHECK(hipFree(d_reference));
    CHECK(hipFree(d_sc_mat_list));
    CHECK(hipFree(d_dir_mat_list));
    CHECK(hipFree(d_res));
    CHECK(hipFree(d_simple_rev_cigar));

	free(h_query);
	free(h_reference);
	free(h_sc_mat);
	free(h_dir_mat);
	free(h_res);
	free(h_simple_rev_cigar);

    return 0;
}