
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define S_LEN 512
#define N 1000

#define PEN_INS -2
#define PEN_DEL -2
#define PEN_MATCH 1
#define PEN_MISMATCH -1

#define VAL 0
#define J   1

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

double get_time() // function that returns the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

__device__ int max4(int n1, int n2, int n3, int n4)
{
	int tmp1, tmp2;
	tmp1 = n1 > n2 ? n1 : n2;
	tmp2 = n3 > n4 ? n3 : n4;
	tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
	return tmp1;
}

__device__ void backtrace(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
	int n;
	for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
	{
		int dir = dir_mat[i][j];
		if (dir == 1 || dir == 2)
		{
			i--;
			j--;
		}
		else if (dir == 3)
			i--;
		else if (dir == 4)
			j--;

		simple_rev_cigar[n] = dir;
	}
}

__device__ int setDirScore(char * query, char * reference, int ** sc_mat, char ** dir_mat, int x, int y) {
	// penalities (initialized above)
	// 	ins = -2
	// 	del = -2
	// 	match = 1
	// 	mismatch = -1 

	// compare the sequences characters
	int comparison = (query[x - 1] == reference[y - 1]) ? PEN_MATCH : PEN_MISMATCH;
	// compute the cell knowing the comparison resultS
	int score = max4(sc_mat[x - 1][y - 1] + comparison, sc_mat[x - 1][y] + PEN_DEL, sc_mat[x][y- 1] + PEN_INS, 0);

	// directions
	// 	1 -> up + left
	// 	2 -> up + left
	// 	3 -> left
	// 	4 -> up
	//  0 -> none.
	char dir;

	if (score == (sc_mat[x - 1][y - 1] + comparison))
		dir = comparison == PEN_MATCH ? 1 : 2;
	else if (score == (sc_mat[x - 1][y] + PEN_DEL))
		dir = 3;
	else if (score == (sc_mat[x][y - 1] + PEN_INS))
		dir = 4;
	else
		dir = 0;

	dir_mat[x][y] = dir;
	sc_mat[x][y] = score;

	return score;
}

__global__ void sw_GPU(char ** query, char ** reference, int *** sc_mat_list, char *** dir_mat_list, int * res, char ** simple_rev_cigar) {    
    int n = blockIdx.x;

	int ** sc_mat = sc_mat_list[n];
    char ** dir_mat = dir_mat_list[n];

    int i, progress;
    int score;
    int max, maxi, maxj;

    // clock_t start, end;

    __shared__ int max_supp[S_LEN][2];

    for(i=0; i<S_LEN; i++) {
        sc_mat[i][threadIdx.x] = 0;
        dir_mat[i][threadIdx.x] = 0;
    }
    // Additional instruction for the remaining row
    if(threadIdx.x == S_LEN-1) {
        sc_mat[i][S_LEN] = 0;
        dir_mat[i][S_LEN] = 0;
    }

    progress = 1;
    for(i=1; i<S_LEN*2; i++) {
        if(threadIdx.x < i && progress <= S_LEN) {
            score = setDirScore(query[n], reference[n], sc_mat, dir_mat, threadIdx.x+1, progress);
            progress++;
        }
        __syncthreads();
    }


    max_supp[threadIdx.x][VAL] = sc_mat[threadIdx.x][1];
    max_supp[threadIdx.x][J] = 1;
    for(i=2; i<S_LEN; i++) {
        if(sc_mat[threadIdx.x][i] > max_supp[threadIdx.x][VAL]) {
            max_supp[threadIdx.x][VAL] = sc_mat[threadIdx.x][i];
            max_supp[threadIdx.x][J] = i;
        }
    }
        
    __syncthreads();        
    if(threadIdx.x == 0) {
        max = max_supp[0][VAL];
        maxi = threadIdx.x;
        maxj = max_supp[0][J];
        for(i=1; i<S_LEN; i++) {
            if(max_supp[i][VAL] > max) {
                max = max_supp[i][VAL];
                maxi = i;
                maxj = max_supp[i][J];
            }
        }    

        res[n] = sc_mat[maxi][maxj];
        backtrace(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN*2);

        // if(blockIdx.x == 0) {
        //     printf("Score matrix\t\tDirection matrix\n");
        //     for(i=0; i<S_LEN; i++) {
        //         for(j=0; j<S_LEN+1; j++) {
        //             printf("%d ", sc_mat[i][j]);
        //         }
        //         printf("\t");
        //         for(j=0; j<S_LEN+1; j++) {
        //             printf("%d ", dir_mat[i][j]);
        //         }
        //         printf("\n");
        //     }
        //     printf("max = %d, maxi = %d, maxj = %d\n", max, maxi, maxj);
        // }
    }
}


__host__ int max4_CPU(int n1, int n2, int n3, int n4)
{
	int tmp1, tmp2;
	tmp1 = n1 > n2 ? n1 : n2;
	tmp2 = n3 > n4 ? n3 : n4;
	tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
	return tmp1;
}

__host__ void backtrace_CPU(char *simple_rev_cigar, char **dir_mat, int i, int j, int max_cigar_len)
{
	int n;
	for (n = 0; n < max_cigar_len && dir_mat[i][j] != 0; n++)
	{
		int dir = dir_mat[i][j];
		if (dir == 1 || dir == 2)
		{
			i--;
			j--;
		}
		else if (dir == 3)
			i--;
		else if (dir == 4)
			j--;

		simple_rev_cigar[n] = dir;
	}
}

__host__ void sw_CPU(char ** query, char ** reference, int ** sc_mat, char ** dir_mat, int * res, char ** simple_rev_cigar) {
	int ins = -2, del = -2, match = 1, mismatch = -1; // penalties
	
	for (int n = 0; n < N; n++)
	{
		int max = ins; // in sw all scores of the alignment are >= 0, so this will be for sure changed
		int maxi, maxj;
		// initialize the scoring matrix and direction matrix to 0
		for (int i = 0; i < S_LEN + 1; i++)
		{
			for (int j = 0; j < S_LEN + 1; j++)
			{
				sc_mat[i][j] = 0;
				dir_mat[i][j] = 0;
			}
		}
		// compute the alignment
		for (int i = 1; i < S_LEN; i++)
		{
			for (int j = 1; j < S_LEN; j++)
			{
				// compare the sequences characters
				int comparison = (query[n][i - 1] == reference[n][j - 1]) ? match : mismatch;
				// compute the cell knowing the comparison result
				int tmp = max4_CPU(sc_mat[i - 1][j - 1] + comparison, sc_mat[i - 1][j] + del, sc_mat[i][j - 1] + ins, 0);
				char dir;

				if (tmp == (sc_mat[i - 1][j - 1] + comparison))
					dir = comparison == match ? 1 : 2;
				else if (tmp == (sc_mat[i - 1][j] + del))
					dir = 3;
				else if (tmp == (sc_mat[i][j - 1] + ins))
					dir = 4;
				else
					dir = 0;

				dir_mat[i][j] = dir;
				sc_mat[i][j] = tmp;

				if (tmp > max)
				{
					max = tmp;
					maxi = i;
					maxj = j;
				}
			}
		}
		res[n] = sc_mat[maxi][maxj];
		backtrace_CPU(simple_rev_cigar[n], dir_mat, maxi, maxj, S_LEN * 2);

        // if(n == 0) {
        //     printf("Score matrix\t\tDirection matrix\n");
        //     for(int i=0; i<S_LEN; i++) {
        //         for(int j=0; j<S_LEN+1; j++) {
        //             printf("%d ", sc_mat[i][j]);
        //         }
        //         printf("\t");
        //         for(int j=0; j<S_LEN+1; j++) {
        //             printf("%d ", dir_mat[i][j]);
        //         }
        //         printf("\n");
        //     }
        //     printf("max = %d, maxi = %d, maxj = %d\n", max, maxi, maxj);
        // }
    }
}

int main(int argc, char * argv[]) {
	srand(time(NULL)); 
    char alphabet[5] = {'A', 'C', 'G', 'T', 'N'};
    double time_start, time_stop;

    // Host memory allocation and initialization for sequences (randomly generated)
    char **h_query = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_query[i] = (char *)malloc(S_LEN * sizeof(char));
	char **h_reference = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_reference[i] = (char *)malloc(S_LEN * sizeof(char));

	int **h_sc_mat = (int **)malloc((S_LEN + 1) * sizeof(int *));
	for (int i = 0; i < (S_LEN + 1); i++)
		h_sc_mat[i] = (int *)malloc((S_LEN + 1) * sizeof(int));
	char **h_dir_mat = (char **)malloc((S_LEN + 1) * sizeof(char *));
	for (int i = 0; i < (S_LEN + 1); i++)
		h_dir_mat[i] = (char *)malloc((S_LEN + 1) * sizeof(char));

	int *h_res = (int *)malloc(N * sizeof(int));
	char **h_simple_rev_cigar = (char **)malloc(N * sizeof(char *));
	for (int i = 0; i < N; i++)
		h_simple_rev_cigar[i] = (char *)malloc(S_LEN * 2 * sizeof(char));


	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < S_LEN; j++)
		{
			h_query[i][j] = alphabet[rand() % 5];
			h_reference[i][j] = alphabet[rand() % 5];
		}
	}

    // Device memory allocation and transfer of input data
    char **d_query;
    char **d_query_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_query,  N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_query_ptrs[i]), S_LEN*sizeof(char));
        hipMemcpy (d_query_ptrs[i], h_query[i], S_LEN*sizeof(char), hipMemcpyHostToDevice);
    }
    hipMemcpy (d_query, d_query_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);

    char **d_reference;
    char **d_reference_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_reference,  N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_reference_ptrs[i]), S_LEN*sizeof(char));
        hipMemcpy (d_reference_ptrs[i], h_reference[i], S_LEN*sizeof(char), hipMemcpyHostToDevice);
    }
    hipMemcpy (d_reference, d_reference_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);
	
	int **d_sc_mat;
    int **d_sc_mat_ptrs;
	int ***d_sc_mat_list;
	int ***d_sc_mat_list_ptrs = (int ***)malloc(N * sizeof(int **));
    hipMalloc((void***)&d_sc_mat_list, N * sizeof(int **));
	for(int j=0; j<N; j++) {
		d_sc_mat_ptrs = (int **)malloc((S_LEN+1) * sizeof(int *));    
		hipMalloc((void***)&d_sc_mat, (S_LEN+1) * sizeof(int *));
		for(int i=0; i<(S_LEN+1); i++) {
			hipMalloc((void**) &(d_sc_mat_ptrs[i]), (S_LEN+1)*sizeof(int));
		}
		hipMemcpy (d_sc_mat, d_sc_mat_ptrs, (S_LEN+1)*sizeof(int *), hipMemcpyHostToDevice);
		d_sc_mat_list_ptrs[j] = d_sc_mat;
	}
	hipMemcpy (d_sc_mat_list, d_sc_mat_list_ptrs, N*sizeof(int **), hipMemcpyHostToDevice);

	char **d_dir_mat;
    char **d_dir_mat_ptrs;
	char ***d_dir_mat_list;
	char ***d_dir_mat_list_ptrs = (char ***)malloc(N * sizeof(char **));
    hipMalloc((void***)&d_dir_mat_list, N * sizeof(char **));
	for(int j=0; j<N; j++) {
		d_dir_mat_ptrs = (char **)malloc((S_LEN+1) * sizeof(char *));    
		hipMalloc((void***)&d_dir_mat, (S_LEN+1) * sizeof(char *));
		for(int i=0; i<(S_LEN+1); i++) {
			hipMalloc((void**) &(d_dir_mat_ptrs[i]), (S_LEN+1)*sizeof(char));
		}
		hipMemcpy (d_dir_mat, d_dir_mat_ptrs, (S_LEN+1)*sizeof(char *), hipMemcpyHostToDevice);
		d_dir_mat_list_ptrs[j] = d_dir_mat;
	}
	hipMemcpy (d_dir_mat_list, d_dir_mat_list_ptrs, N*sizeof(char **), hipMemcpyHostToDevice);

    int * d_res;
    hipMalloc((void**) &d_res, N*sizeof(int));

    char **d_simple_rev_cigar;
    char **d_simple_rev_cigar_ptrs = (char **)malloc(N * sizeof(char *));    
    hipMalloc((void***)&d_simple_rev_cigar, N * sizeof(char *));
    for(int i=0; i<N; i++) {
        hipMalloc((void**) &(d_simple_rev_cigar_ptrs[i]), S_LEN * 2 * sizeof(char));
    }
    hipMemcpy (d_simple_rev_cigar, d_simple_rev_cigar_ptrs, N*sizeof(char *), hipMemcpyHostToDevice);

    // Blocks and threads schema for GPU execution
    dim3 blocksPerGrid(N, 1, 1);
    dim3 threadsPerBlock(S_LEN, 1, 1);

    // Execution on GPU started
    time_start = get_time();

    sw_GPU<<<blocksPerGrid, threadsPerBlock>>>(d_query, d_reference, d_sc_mat_list, d_dir_mat_list, d_res, d_simple_rev_cigar);
    CHECK_KERNELCALL();
	// The following function adds A LOT of latency! Don't use it for benchmarking purposes
    // cudaDeviceSynchronize();

    // Execution on GPU ended
    time_stop = get_time();
    printf("GPU Execution time: %.10f\n", time_stop-time_start);

	// Transfer data to host
	// int * gpu_res = (int *) malloc(sizeof(int)*N);
	int gpu_res[N];
	char gpu_simple_rev_cigar[N][S_LEN*2];
	char * tmp_pointer;

	hipMemcpy((void *) gpu_res, d_res, sizeof(int)*N, hipMemcpyDeviceToHost);
	for(int i=0; i<N; i++) {
		tmp_pointer = d_simple_rev_cigar_ptrs[i];
		hipMemcpy((void *) gpu_simple_rev_cigar[i], tmp_pointer, sizeof(char)*S_LEN*2, hipMemcpyDeviceToHost);
	}

    // Execution on CPU started
    time_start = get_time();

	sw_CPU(h_query, h_reference, h_sc_mat, h_dir_mat, h_res, h_simple_rev_cigar);
    
	// Execution on CPU ended
    time_stop = get_time();
    printf("CPU Execution time: %.10f [!]\n", time_stop-time_start);

	// Check if results are consistent
	// "Results" validation
	int ok, i, j;
	ok = 1;
	for(i=0; (i<N && ok); i++) {
		ok = gpu_res[i] == h_res[i];
	}

	if(ok)
		printf("[OK]\t'results' is consistent\n");
	else
		printf("[ERR]\t'results' is inconsistent\n");

	ok = 1;
	for(i=0; (i<N && ok); i++) {
		for(j=0; (j<S_LEN*2 && ok); j++) {
			ok = gpu_simple_rev_cigar[i][j] == h_simple_rev_cigar[i][j];
		}
	}
	i--;
	if(ok)
		printf("[OK]\t'rev_cigar' is consistent\n");
	else{
		printf("[ERR]\t'rev_cigar' is inconsistent [err on: %d]\n", i);
	}


	// Deallocation of memory
	CHECK(hipFree(d_query));
    CHECK(hipFree(d_reference));
    CHECK(hipFree(d_sc_mat_list));
    CHECK(hipFree(d_dir_mat_list));
    CHECK(hipFree(d_res));
    CHECK(hipFree(d_simple_rev_cigar));

	free(h_query);
	free(h_reference);
	free(h_sc_mat);
	free(h_dir_mat);
	free(h_res);
	free(h_simple_rev_cigar);

    return 0;
}